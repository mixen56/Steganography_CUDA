// nvcc -c libhidecuda.cu -o libhidecuda.o

#include <iostream>
#include <fstream>
#include <string>
#include <filesystem>

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


using namespace std;
// GLOBAL VARS
// -----------------------------------------------------------------------
// global write step in all frames
__device__ const int META_STEP = 10;

// meta of 0 frame (bits indexes): coordinates of step
// 10 - 100
__device__ const int STEP_START  = 10;
__device__ const int STEP_MAXLEN = 10;

// meta of 0 frame (bits indexes): coordinates of file name
// 210 - 500
__device__ const int CUDA_FILENAME_START  = 210;
__device__ const int FILENAME_MAXLEN = 30;

// meta of 0 frame (bits indexes): coordinates of number of used frames
// 610 - 700
__device__ const int USEDFRAMES_START  = 610;
__device__ const int USEDFRAMES_MAXLEN = 10;

// meta of others frames: mark
__device__ const int MARK_START  = 10;
__device__ const int MARK_STOP   = 50;
__device__ const int MARK_MAXLEN = 5;
__device__ char MARK[] = "abcde";

// meta of others frames: number of symbols in frame
__device__ const int NUMSYBMOLS_START    = 110;
__device__ const int NUMSYBMOLS_STOP     = 200;
__device__ const int NUMSYBMOLS_MAXLEN   = 10;

// meta of others frames: symbols
__device__ const int SYMBOLS_START   = 1000;
__device__ int SYMBOLS_STOP;
__device__ int SYMBOLS_MAXLEN;

// minimal number of symbols in frame
__device__ int SYMBOLS_FRAME_MIN = 10;

__device__ unsigned char SYMBOLS_OUT='*';
// -----------------------------------------------------------------------


__global__
void cuda_kernel_hide(unsigned char *DATA_CUDA, size_t FrameSize,
                 char *FILE_CUDA, size_t FileSize,
                 size_t SymbolNum, char *SymbolNumStr, size_t SymbolNumStrSize, size_t Step, int SYMBOLS_STOP) {
    size_t frame_index = blockIdx.x * FrameSize;                        // start of frame
    size_t file_index_start = blockIdx.x * SymbolNum;                   // start of file part

    size_t IterMeta;            // iterator index
    unsigned char write_char;   // char to write into frame
    char *FirstChar, *LastChar;  // start and stop of writing

    // write mark
    // -----------------------------------------------------------------------
    // cuda_write_data_to_frame(0, MARK, 5, DATA_CUDA, frame_index, 10, 50, 10);
    IterMeta = MARK_START + frame_index;
    FirstChar = MARK;
    LastChar  = FirstChar + MARK_MAXLEN;

    // write info
    for (char *letter = FirstChar; letter < LastChar; letter++ ) {
        write_char = static_cast<unsigned char>(*letter);
        DATA_CUDA[IterMeta] = write_char;
        IterMeta += META_STEP;
    }
    // -----------------------------------------------------------------------

    // write number of symbols
    // -----------------------------------------------------------------------
    // cuda_write_to_frame(SymbolNumStr, SymbolNumStrSize, DATA_CUDA, frame_index, 110, 10);
    IterMeta = NUMSYBMOLS_START + frame_index;
    LastChar = SymbolNumStr + SymbolNumStrSize;  // last symbol of metainfo

    // write info
    for (char *letter = SymbolNumStr; letter < LastChar; letter++ ) {
        write_char = static_cast<unsigned char>(*letter);
        DATA_CUDA[IterMeta] = *letter;
        IterMeta += META_STEP;
    }

    // write end of info
    for (int i = IterMeta; i < NUMSYBMOLS_START + frame_index + NUMSYBMOLS_MAXLEN * META_STEP; i+=META_STEP) {
        write_char = static_cast<unsigned char>('*');
        DATA_CUDA[IterMeta] = write_char;
    }
    // -----------------------------------------------------------------------

    // writer file symbols
    // -----------------------------------------------------------------------
    // cuda_write_data_to_frame(file_index_start, FILE_CUDA, SymbolNum, DATA_CUDA, frame_index, 1000, SYMBOLS_STOP, Step);
    IterMeta = SYMBOLS_START + frame_index;

    // file indexes
    FirstChar = FILE_CUDA + file_index_start;
    LastChar  = FirstChar + SymbolNum;


    // write info
    for (char *letter = FirstChar; letter < LastChar; letter++ ) {
        write_char = static_cast<unsigned char>(*letter);
        DATA_CUDA[IterMeta] = write_char;
        IterMeta += Step;
    }
    // -----------------------------------------------------------------------
}


// edit frames
void cuda_edit_frames(unsigned char *DATA, size_t UsedFramesNumber, size_t FrameSize,
                      string &FileStr, size_t SymbolNum,
                      string SymbolNumStr, size_t Step) {

    // GLOBAL VARS EDIT
    int SYMBOLS_STOP = FrameSize - 10000; // - 10000 just in case

    // copy memory from HOST to GPU
    unsigned char *DATA_CUDA;
    hipMallocManaged(&DATA_CUDA, sizeof(unsigned char) * UsedFramesNumber * FrameSize);
    hipMemcpy(DATA_CUDA, DATA, sizeof(unsigned char) * UsedFramesNumber * FrameSize, hipMemcpyHostToDevice);

    // load file to gpu mem
    char *FILE_CUDA;
    size_t FileSize = FileStr.size();
    hipMallocManaged(&FILE_CUDA, sizeof(char) * FileSize);
    hipMemcpy(FILE_CUDA, FileStr.data(), sizeof(char) * FileSize, hipMemcpyHostToDevice);

    // prepare SymbolNumStr as c string
    char *SymbolNumStrC;
    size_t SymbolNumStrCSize = SymbolNumStr.size();
    hipMallocManaged(&SymbolNumStrC, sizeof(char) * SymbolNumStrCSize);
    hipMemcpy(SymbolNumStrC, SymbolNumStr.data(), sizeof(char) * SymbolNumStrCSize, hipMemcpyHostToDevice);

    cuda_kernel_hide<<<UsedFramesNumber, 1>>>(DATA_CUDA, FrameSize, FILE_CUDA, FileSize, SymbolNum, SymbolNumStrC, SymbolNumStrCSize, Step, SYMBOLS_STOP);
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_kernel!\n", cudaStatus);
    }

    // copy memory from GPU to HOST
    hipMemcpy(DATA, DATA_CUDA, sizeof(unsigned char) * UsedFramesNumber * FrameSize, hipMemcpyDeviceToHost);

    hipFree(DATA_CUDA);
    hipFree(FILE_CUDA);
}


__global__
void cuda_kernel_dishide(unsigned char *DATA_CUDA, size_t FrameSize,
                 char *FILE_CUDA, size_t FileSize,
                 size_t SymbolNum, size_t Step, int SYMBOLS_STOP) {
    size_t frame_index = blockIdx.x * FrameSize;                        // start of frame
    size_t file_index_start = blockIdx.x * SymbolNum;                   // pointer to current file index

    unsigned char get_char;     // get char from frame
    char ready_char;            // get char from frame and convert to char
    size_t j;                      // iter index

    // get MARK and check existance
    // -----------------------------------------------------------------------
    char mark[MARK_MAXLEN + 1];
    j = 0;

    for (int i = MARK_START + frame_index; i <= MARK_STOP + frame_index; i+=META_STEP) {
        get_char = DATA_CUDA[i];
        ready_char = static_cast<char>(get_char);

        mark[j] = ready_char;
        j++;
    }
    mark[MARK_MAXLEN] = '\0';

    // if no mark
    bool go = true;
    for (j = 0; j < MARK_MAXLEN; j++) {
        if (mark[j] != MARK[j]) {
            printf ("CUDA %ld: COMAPARE (%c %c)\n", blockIdx.x, mark[j], MARK[j]);
            go = false;
            break;
        }
    }
    // -----------------------------------------------------------------------

    // get symbol num
    // -----------------------------------------------------------------------
    if (go) {
        char numsymbols_str[NUMSYBMOLS_MAXLEN + 1];

        j = 0;
        for (int i = NUMSYBMOLS_START + frame_index; i < NUMSYBMOLS_STOP + frame_index; i+=META_STEP) {
            get_char = DATA_CUDA[i];;
            ready_char = static_cast<char>(get_char);
            if (ready_char == SYMBOLS_OUT)
                break;

            numsymbols_str[j] = ready_char;
            j++;
        }
        numsymbols_str[j] = '\0';

        // size_t numsymbols = stoi(numsymbols_str);
        // my stoi
        int numsymbols = 0;
        char ch;
        int cur_power = 1;

        for (int i = j - 1; i >= 0; i--) {
            ch = numsymbols_str[i];
            switch (ch) {
                case '0':
                    numsymbols = numsymbols + 0 * cur_power;
                    break;
                case '1':
                    numsymbols = numsymbols + 1 * cur_power;
                    break;
                case '2':
                    numsymbols = numsymbols + 2 * cur_power;
                    break;
                case '3':
                    numsymbols = numsymbols + 3 * cur_power;
                    break;
                case '4':
                    numsymbols = numsymbols + 4 * cur_power;
                    break;
                case '5':
                    numsymbols = numsymbols + 5 * cur_power;
                    break;
                case '6':
                    numsymbols = numsymbols + 6 * cur_power;
                    break;
                case '7':
                    numsymbols = numsymbols + 7 * cur_power;
                    break;
                case '8':
                    numsymbols = numsymbols + 8 * cur_power;
                    break;
                case '9':
                    numsymbols = numsymbols + 9 * cur_power;
                    break;
            }
            cur_power*=10;  // increase power
        }
    // -----------------------------------------------------------------------

    // get data from frame and write to file
    // -----------------------------------------------------------------------
    //if (go) {
        j = file_index_start;
        for (int i = SYMBOLS_START + frame_index; i <= SYMBOLS_START + frame_index + numsymbols * Step - 1; i+=Step) {
            get_char = DATA_CUDA[i];
            ready_char = static_cast<char>(get_char);

            FILE_CUDA[j] = ready_char;
            j++;
        }
    }
    // last frame
    if (gridDim.x == (blockIdx.x + 1)) {
        FILE_CUDA[j] = '\0';
    }
    // -----------------------------------------------------------------------
}


// fill FILE
void cuda_read_from_frames(unsigned char *DATA, size_t UsedFramesNumber, size_t FrameSize,
                      char *FileStr, size_t SymbolNum,
                      size_t Step) {

    // GLOBAL VARS EDIT
    int SYMBOLS_STOP = FrameSize - 10000; // - 10000 just in case

    // copy memory from HOST to GPU
    unsigned char *DATA_CUDA;
    hipMallocManaged(&DATA_CUDA, sizeof(unsigned char) * UsedFramesNumber  * FrameSize);
    hipMemcpy(DATA_CUDA, DATA, sizeof(unsigned char) * UsedFramesNumber * FrameSize, hipMemcpyHostToDevice);

    // load file to gpu mem
    char *FILE_CUDA;
    size_t FileSize = UsedFramesNumber * SymbolNum;
    hipMallocManaged(&FILE_CUDA, sizeof(char) * FileSize);
    hipMemcpy(FILE_CUDA, FileStr, sizeof(char) * FileSize, hipMemcpyHostToDevice);

    cuda_kernel_dishide<<<UsedFramesNumber, 1>>>(DATA_CUDA, FrameSize, FILE_CUDA, FileSize, SymbolNum, Step, SYMBOLS_STOP);
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_kernel!\n", cudaStatus);
    }

    // copy memory from GPU to HOST
    hipMemcpy(FileStr, FILE_CUDA, sizeof(char) * UsedFramesNumber * SymbolNum, hipMemcpyDeviceToHost);
    // thanks father for memcpy bug
    FileStr[UsedFramesNumber * SymbolNum] = '\0';

    hipFree(DATA_CUDA);
    hipFree(FILE_CUDA);
}
